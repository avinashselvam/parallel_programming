
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void shift(int * g){

    int i = threadIdx.x;

    __shared__ int array[128];

    array[i] = i;
    __syncthreads();

    if(i<127){
        int temp = array[i + 1];
        __syncthreads();
        
        array[i] = temp;
        __syncthreads();
    }

    g[i] = array[i];
    __syncthreads(); // not really necessary as no further operations
}

// helper function

void print_array(int * h){
    for(int i=0; i < 128; i=i+1){
        printf("%d ", h[i]);
    }
    printf("\n");
}

int main(){

    // array on host memory
    int h_array[128];
    const int ARRAY_BYTES = sizeof(int)*128;

    // array on Device global memory
    int * d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);

    // kernel call
    shift<<<1,128>>>(d_array);

    // get results from Device global to host
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // see results
    print_array(h_array);

    hipFree(d_array);
    
    return 0;
}