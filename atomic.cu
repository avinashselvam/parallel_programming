#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gputimer.h"

#define BLOCK_SIZE 1000
#define ARRAY_SIZE 100
#define NUM_THREADS 1000

__global__ void naive_add(int * g){

    int i = blockIdx.x*blockDim.x + threadIdx.x;

    i = i % ARRAY_SIZE;
    g[i] = g[i] + 1;

}

__global__ void atomic_add(int * g){

    int i = blockIdx.x*blockDim.x + threadIdx.x;

    i = i % ARRAY_SIZE;
    atomicAdd(& g[i], 1);

}

void print_array(int * h){
    for(int i=0; i < ARRAY_SIZE; i=i+1){
        printf("%d ", h[i]);
    }
    printf("\n");
}

int main(int argc, char **argv){
    
    GpuTimer timer;

    // Declaring and allocating host memory
    int h_array[ARRAY_SIZE];
    int ARRAY_BYTES = sizeof(int)*ARRAY_SIZE;

    // Declaring, allocating and assign zero to GPU memory
    int *d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES);

    // first kernel
    timer.Start();
    naive_add<<<BLOCK_SIZE, NUM_THREADS>>>(d_array);
    timer.Stop();

    // back to host and print it
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array);
    hipMemset((void *) d_array, 0, ARRAY_BYTES);    
    printf("Time elapsed using naive addition = %g ms\n", timer.Elapsed());

     // second kernel
     timer.Start();
     atomic_add<<<BLOCK_SIZE, NUM_THREADS>>>(d_array);
     timer.Stop();
 
     // back to host and print it
     hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
     print_array(h_array);
     printf("Time elapsed using atomic addition = %g ms\n", timer.Elapsed());

    hipFree(d_array);
    return 0;

}