
#include <hip/hip_runtime.h>
#include<stdio.h>

#define ARRAY_SIZE 128*128
#define NUM_THREADS 128
#define BLOCK_SIZE 128

__global__ void reduce(float* d_out, float* d_in){

    int global_id = blockDim.x*blockIdx.x + threadIdx.x;
    int local_id = threadIdx.x;
    
    //extern __shared__ float s_in[];

    for(unsigned int s = blockDim.x/2; s > 0; s>>=1){
        if(local_id < s) d_in[global_id] += d_in[global_id + s];
        __syncthreads();
    }

    if(local_id==0) d_out[blockIdx.x] = d_in[global_id];
    __syncthreads();

}

int main(){

    float *d_in, *d_out, *d_final;
    const int ARRAY_BYTES = ARRAY_SIZE*sizeof(float);
    hipMalloc((void**) &d_in, ARRAY_BYTES);
    hipMalloc((void**) &d_out, ARRAY_BYTES/NUM_THREADS);
    hipMalloc((void**) &d_final, sizeof(float));
    

    float h_array[ARRAY_SIZE];

    for(int i=0; i < ARRAY_SIZE; i+=1){
        h_array[i] = 1;
    }

    float sum;

    hipMemcpy(d_in, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);

    reduce<<<BLOCK_SIZE, NUM_THREADS>>>(d_out, d_in);
    reduce<<<1, NUM_THREADS>>>(d_final, d_out);

    hipMemcpy(&sum, d_out, sizeof(float), hipMemcpyDeviceToHost);

    printf("%f",sum);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_final);


    return 0;
}