#include "hip/hip_runtime.h"
#include<stdio.h>

#define NUM_THREADS 100
#define BLOCK_SIZE 100 

__global__ void blur(int * g, int * img, int * filter){
    int i = threadIdx.x;
    
}

int main(int argv, char* argc){
    
    int img[100][100];
    int filter[4][4] = {1, 1, 1, 1, 1, 3, 3, 1, 1, 3, 3, 1, 1, 1, 1, 1};

    const int ARRAY_BYTES = sizeof(int)*100*100;
    
    // creating half black half white image

    for(int i=0; i<100; i=i+1){
        for(int j=0; j<50; j=j+1){
            img[i][j] = 0;
        }
    }

    for(int i=0; i<100; i=i+1){
        for(int j=50; j<100; j=j+1){
            img[i][j] = 255;
        }
    }

    // declaring and allocating GPU memory

    int* d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES);

    int* d_img;
    hipMalloc((void **) &d_img, ARRAY_BYTES);
    hipMemcpy(d_img, img, ARRAY_BYTES, hipMemcpyHostToDevice);

    blur<<<BLOCK_SIZE, NUM_THREADS>>>(d_array, d_img, filter);






    return 0;
}